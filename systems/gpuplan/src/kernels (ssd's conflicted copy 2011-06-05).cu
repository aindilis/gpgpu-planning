#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "GPUCheck.h"

#include "defines.h"

#undef get16bits
#if (defined(__GNUC__) && defined(__i386__)) || defined(__WATCOMC__) \
	  || defined(_MSC_VER) || defined (__BORLANDC__) || defined (__TURBOC__)
#define get16bits(d) (*((const uint16_t *) (d)))
#endif

#if !defined (get16bits)
#define get16bits(d) ((((unsigned int)(((const unsigned char *)(d))[1])) << 8)\
		                       +(unsigned int)(((const unsigned char *)(d))[0]) )
#endif

#define tId  (blockIdx.x * blockDim.x + threadIdx.x)

#define constantSize 25600*1024


//__constant__ POSTFIX_EFFECTS_TYPE constant[63*1024/4];
__constant__ POSTFIX_EFFECTS_TYPE constant[constantSize/4];
__constant__ unsigned int sizetTransitions_GPU;
__constant__ unsigned int metricTransitionGPU;
__constant__ POSTFIX_EFFECTS_TYPE pr; //postfix representation on the GPU
__constant__ POSTFIX_EFFECTS_TYPE pg; //postfix guards on the gpu
unsigned int cpu_pr;
unsigned int cpu_pg;
__device__ POSTFIX_EFFECTS_TYPE * pe; //postfix effects on the gpu
__device__ unsigned int * to; //transition offsetts on the gpu
__constant__ unsigned int checkSizeGPU;
__constant__ unsigned int stateSizeGPU;
__device__ unsigned int * preconditions_gpu;

__global__ void initKernel(POSTFIX_EFFECTS_TYPE pg_new, POSTFIX_EFFECTS_TYPE * pe_new, unsigned int pr_new, unsigned int * to_new,unsigned int * preconditions_gpu_new) {
	//pg = pg_new;
	pe = pe_new;
//	pr = pr_new;
	to = to_new;
	preconditions_gpu = preconditions_gpu_new;
}

__device__ void memcpyGPU(void * dst, const void * src, size_t bytes) {
	for (int i = 0; i < bytes; i++)
		((char *)dst)[i] = ((char* )src)[i];
}
	
__device__ void getValue(int * value, unsigned char * state, unsigned int * type_stack, int * offset_stack, unsigned int depth){
	*value = 0;
	if (type_stack[depth] == CONSTANT) *value = offset_stack[depth];
	else if (type_stack[depth] == VARIABLE_BYTE) {
		*value= state[offset_stack[depth]]; //byte
	}
	else if (type_stack[depth] == VARIABLE_INT) {
		memcpyGPU(value,&state[offset_stack[depth]],4);  //integer
	}
	else {
		*value= state[offset_stack[depth]>>3] & ((unsigned char) 128 >> (offset_stack[depth]&7));
	}
	
}

__device__ unsigned int evaluatePostfixGPU(unsigned char  * state,POSTFIX_EFFECTS_TYPE * postfix, unsigned int length, bool checkonly){
	unsigned int depth = 0;
	unsigned int postpos = 0;
	unsigned int type_stack[20];
	int offset_stack[20];
	int var1, var2 =0;

	while (postpos < length) {
		if (VARIABLE_BIT <= postfix[postpos] && postfix[postpos] <= CONSTANT) {
			type_stack[depth] = postfix[postpos];
			offset_stack[depth] = postfix[postpos+1];
			depth++;
			postpos += 2;
			continue;
		}
		getValue(&var2,state,type_stack,offset_stack,depth-1);
		if (postfix[postpos] == ARRAY_BYTE) {
			type_stack[depth-1] = VARIABLE_BYTE;
			offset_stack[depth-1] = postfix[postpos+1] + var2; //array offset + variable on stack
			//printf("\n array type_stack[%i] = VARIABLE_BYTE offset_stack = %i \n",depth-1,offset_stack[depth-1]);
			postpos += 2;
			continue;
		}
		if (postfix[postpos] == ARRAY_INT) {
			type_stack[depth-1] = VARIABLE_INT;
			offset_stack[depth-1] = postfix[postpos+1] + (var2*4); //array offset + variable on stack
			//printf("\n array type_stack[%i] = VARIABLE_BYTE offset_stack = %i \n",depth-1,offset_stack[depth-1]);
			postpos += 2;
			continue;
		}
		else if (postfix[postpos] == ASSIGN){
			if (type_stack[depth-2] == VARIABLE_INT) {
				memcpyGPU(&state[offset_stack[depth-2]] , &var2,4);
			}
			else if (type_stack[depth-2] == VARIABLE_BYTE) {
				state[offset_stack[depth-2]] = var2;
			}
			else if (var2)
				state[offset_stack[depth-2]>>3] += ((unsigned char)128 >> (offset_stack[depth-2]&7)); 
			else 
				state[offset_stack[depth-2]>>3] &= ~((unsigned char) 128 >> (offset_stack[depth-2]&7));;
			depth--;
			postpos++;
			continue;
		}
		type_stack[depth-1] = CONSTANT;
		if (postfix[postpos] == NOT) {
			offset_stack[depth-1] = not(var2); //array offset + variable on stack
			postpos += 1;
			continue;
		}
		else if (postfix[postpos] == NEG) {
			offset_stack[depth-1] = -1 * var2; //array offset + variable on stack
			postpos += 1;
			continue;
		}

		getValue(&var1,state,type_stack,offset_stack,depth-2);
		type_stack[depth-2] = CONSTANT;
		if (postfix[postpos] == AND){
			offset_stack[depth-2] = (var1 && var2) ;
		}
		else if (postfix[postpos] == PLUS){
			offset_stack[depth-2] = var1 + var2;
		}
		else if (postfix[postpos] == MINUS){
			offset_stack[depth-2] = var1 - var2;
			//printf("\n storing  %i - %i = %i at pos %i \n",var1, var2,offset_stack[depth-2],depth-2);
		}
		else if (postfix[postpos] == MULT){
			offset_stack[depth-2] = var1 * var2;
			//printf("\n storing  %i * %i = %i at pos %i \n",var1, var2,offset_stack[depth-2],depth-2);
		}
		else if (postfix[postpos] == MOD){
			offset_stack[depth-2] = var1 % var2;
			//printf("\n storing %i - %i at pos %i \n",sum1, sum2,depth-2);
		}
		else if (postfix[postpos] == EQ){
			offset_stack[depth-2] = (var1 == var2) ;
			//printf("\n storing  %i == %i = %i at pos %i \n",var1, var2,offset_stack[depth-2],depth-2);
		}
		else if (postfix[postpos] == NEQ){
			offset_stack[depth-2] = (var1 != var2) ;
			//printf("\n storing  %i != %i at pos %i \n",var1, var2,depth-2);
		}
		else if (postfix[postpos] == LE){
			offset_stack[depth-2] = (var1 <= var2) ;
		}
		else if (postfix[postpos] == LESS){
			offset_stack[depth-2] = (var1 < var2) ;
		}
		else if (postfix[postpos] == GREATER){
			offset_stack[depth-2] = (var1 > var2) ;
		}
		else if (postfix[postpos] == GE){
			offset_stack[depth-2] = (var1 >= var2) ;
		}
		else if (postfix[postpos] == OR){
			offset_stack[depth-2] = (var1 || var2) ;
		}
		else if (postfix[postpos] == BINAND){
			offset_stack[depth-2] = (var1 & var2) ;
			//printf("\n storing  %i & %i = %i at pos %i \n",var1, var2,offset_stack[depth-2],depth-2);
		}
		else if (postfix[postpos] == BINOR){
			offset_stack[depth-2] = (var1 | var2) ;
		}
		depth--;
		postpos++;
	}
	if (checkonly) return offset_stack[0];
	else return true;
}

/* 
	 pr array
	 1 number of processes (pos 0 in pr)
	 pr[0]*2 for each process pr contains a pair (offset of processstate in statevector, sum of transitions for different states for all processes til now)
	 ps*3 for each processstate a tripple is stored (starting processstate, number of guards, guard offset)
	 (p>0)*tnum*3 skip tripples for transitions of previous states 
 */

 __global__ void checkGuardsKernel(unsigned char * stateset, unsigned int setSize){
	if (tId < setSize) {
		unsigned char state[40];
		//unsigned char * state = &stateset[tId*checkSizeGPU];
		//char enabled[200];
		memcpyGPU((void*)state,(void*)&stateset[tId*checkSizeGPU],stateSizeGPU);
		unsigned char * enabled = &stateset[tId*checkSizeGPU];
		int p;
		for (p = 0; p < checkSizeGPU; p++) enabled[p] =0;
		//memset(enabled,0,checkSizeGPU);
		//for (p = 1; p <= constant[pr+0]; p++) {
		p = constant[pr+0]; //read number of processes
		while(p) { //go through each process
			unsigned int ps = state[constant[pr+1+((p-1)*2)]]; //read the state the process is in
			unsigned int tnum = constant[pr+((p-1)*2)]; //number of transition tripples  to skip 
			unsigned int tmp = ((p-1)>0)*tnum*3; //number of entries to skip using tnum
			unsigned int start_guard = constant[pr+1+(constant[pr+0]*2)+ps*3+tmp]; //number of starting guard
			unsigned int guards = constant[pr+1+(constant[pr+0]*2)+ps*3+1+tmp];
			unsigned int guard_offset = constant[pr+1+(constant[pr+0]*2)+ps*3+2+tmp];
			unsigned int gsize;
			while (guards) { //check each guard of the process
				gsize = constant[pg+guard_offset];
				if (!gsize || evaluatePostfixGPU(state,&constant[pg+guard_offset+1],gsize,true)) {
					enabled[start_guard>>3] += (unsigned char) 128 >> (start_guard &7 );
				}
				else 
					*(enabled+(start_guard>>3)) = enabled[start_guard>>3]; // streamed access to memor :-D
				start_guard++;
				guard_offset += gsize+1;
				guards--;
			}
			p--;
		}
		//__syncthreads();
		//for (p = 0; p < checkSizeGPU; p++) state[p] = enabled[p];
	//	memcpyGPU(state,enabled,checkSizeGPU);
	}
};

/* pe array
	 pe[0] ???
	 pe[1] target process state
	 pe[2] number of effects
	 pe[3] length of first effect
	 pe[4] effect in postfix
	 */
__global__ void enableEffectKernel(unsigned char * stateset, unsigned int setSize){

	if (tId < setSize) {
		unsigned int transition=0; 
		memcpyGPU(&transition, &stateset[(tId+1)*(stateSizeGPU+sizetTransitions_GPU)-sizetTransitions_GPU],sizetTransitions_GPU);
		unsigned char * state = &stateset[tId*(stateSizeGPU+sizetTransitions_GPU)];
		unsigned int effects = pe[to[transition]+2];
		unsigned int effect_length = pe[to[transition]+3];
		unsigned int effect_offset = to[transition]+4;
//	__syncthreads();

		state[pe[to[transition]]] = pe[to[transition]+1]; //set new state for process
//		printf("thread %i tries to enable %i effects in transition %i \n",tId,effects,transition);
		while (effects) {
			//printf("\n evaluating effect starting at %i , length %i ",effect_offset,effect_length);
			if (!evaluatePostfixGPU(state,&pe[effect_offset],effect_length,false)) ;
			effects--;
			effect_offset += effect_length+1;
			effect_length = pe[effect_offset-1];
		}
//		printf("thread %i enabled transition %i \n",tId,transition);
	}
}

__global__ void computeMetricKernel(unsigned char * stateset, unsigned int setSize){
	if (tId < setSize) {
		unsigned char * state = &stateset[tId*(stateSizeGPU+sizetTransitions_GPU)];
		unsigned int transition=metricTransitionGPU; 
		unsigned int effect_length = pe[to[transition]+3];
		unsigned int effect_offset = to[transition]+4;
//	__syncthreads();

			//printf("\n evaluating effect starting at %i , length %i ",effect_offset,effect_length);
		transition = evaluatePostfixGPU(state,&pe[effect_offset],effect_length,true) ;
		memcpyGPU(&stateset[(tId+1)*(stateSizeGPU+sizetTransitions_GPU)-sizetTransitions_GPU],&transition,sizetTransitions_GPU);
	}
}

__global__ void getHashesKernel(unsigned char * stateset, unsigned int setSize){
	if (tId < setSize) {

		unsigned int hash = stateSizeGPU, tmp;
		int rem = stateSizeGPU & 3;
		for ( int i = 0; i < 2; i++ ){
			unsigned char * data = &stateset[tId*(stateSizeGPU+sizetTransitions_GPU)];
			unsigned int len = stateSizeGPU >>2;

			/* Main loop */
			for (;len > 0; len--) {
				hash  += get16bits (data);
				tmp    = (get16bits (data+2) << 11) ^ hash;
				hash   = (hash << 16) ^ tmp;
				data  += 4;
				hash  += hash >> 11;
			}

			/* Handle end cases */
			if (rem) {
				if (rem == 3) {
					hash += get16bits (data);
					hash ^= hash << 16;
					hash ^= data[2] << 18;
					hash += hash >> 11;
				}
				else if (rem == 2) {
					hash += get16bits (data);
					hash ^= hash << 11;
					hash += hash >> 17;
				}
				else if (rem == 1) {
					hash += *data;
					hash ^= hash << 10;
					hash += hash >> 1;
				}
			}

			/* Force "avalanching" of final 127 bits */
			hash ^= hash << 3;
			hash += hash >> 5;
			hash ^= hash << 4;
			hash += hash >> 17;
			hash ^= hash << 25;
			hash += hash >> 6;

			hash %= HASHTABLESIZE;

			memcpyGPU(&stateset[tId*(stateSizeGPU+sizetTransitions_GPU)+i*sizeof(unsigned int)],&hash,sizeof(unsigned int));
		}

	}

}

 __global__ void computeHeuristicKernel(unsigned char * stateset, unsigned int setSize){
	if (tId < setSize) {
		unsigned char state[200];
		unsigned char * orgState = &stateset[tId*(stateSizeGPU+sizetTransitions_GPU)];
		//unsigned char * state = &stateset[tId*checkSizeGPU];
		//char enabled[200];
		memcpyGPU(state,orgState,stateSizeGPU);
		int p;
		//for (p = 1; p <= constant[pr+0]; p++) {
		p = constant[pr+0]; //read number of processes
		while(p) { //go through each process
			unsigned int ps = state[constant[pr+1+((p-1)*2)]]; //read the state the process is in
			unsigned int tnum = constant[pr+((p-1)*2)]; //number of transition tripples  to skip 
			unsigned int tmp = ((p-1)>0)*tnum*3; //number of entries to skip using tnum
			unsigned int start_guard = constant[pr+1+(constant[pr+0]*2)+ps*3+tmp]; //number of starting guard
			unsigned int guards = constant[pr+1+(constant[pr+0]*2)+ps*3+1+tmp];
			unsigned int guard_offset = constant[pr+1+(constant[pr+0]*2)+ps*3+2+tmp];
			unsigned int gsize;
			while (guards) { //check each guard of the process
				gsize = constant[pg+guard_offset];
				if (!gsize || evaluatePostfixGPU(orgState,&constant[pg+guard_offset+1],gsize,true)) {
					//enabled[start_guard>>3] += (unsigned char) 128 >> (start_guard &7 );
				}
				start_guard++;
				guard_offset += gsize+1;
				guards--;
			}
			p--;
		}
		//for (p = 0; p < checkSizeGPU; p++) state[p] = enabled[p];
//		memcpy(state,enabled,checkSizeGPU);
	}
};
