/*
 * GPUCheck.cpp
 *
 *  Created on: 05.01.2010
 *      Author: sulewski
 */
#include "GPUCheck.h"

#include "kernels.cu"

__device__ unsigned char * vram; //pointer to GPU memory


__host__ unsigned int computeGroups(unsigned int setSize) {
	unsigned int groups = setSize/THREADS+1;
	if (groups > 65000) {
		printf("\t %i groups splitting \n",groups);
		groups = 65000;
	}

	return groups;
}

void GPUCheck(myModel * model) { //initialize GPU memories

	printf("Initializing GPU \n");

	hipError_t error = hipMalloc((void **) &vram, VRAM);
	if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %s\n","Error allocating vram asfsdf" ,hipGetErrorString(error)); exit(-1); }
	printf("reserved %i MB (%i bytes) on GPU \n",VRAM/1024/1024,VRAM);

	cpu_pr =0;
	hipMemcpyToSymbol(HIP_SYMBOL("pr"),&cpu_pr,sizeof(unsigned int),0,hipMemcpyHostToDevice);
	//hipMalloc((void **) &pr,model->postfix_representation.size()*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL("constant"),&model->postfix_representation[0],model->postfix_representation.size()*sizeof(int),pr,hipMemcpyHostToDevice);
	printf("Copied postfix_representation %lu bytes to %i \n",(int)model->postfix_representation.size()*sizeof(int),pr);

	
	cpu_pg = cpu_pr+model->postfix_representation.size();
	hipMemcpyToSymbol(HIP_SYMBOL("pg"),&cpu_pg,sizeof(unsigned int),0,hipMemcpyHostToDevice);
	//hipMalloc((void **) &pg,model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE)); 
	hipMemcpyToSymbol(HIP_SYMBOL("constant"),&model->postfix_guards[0],model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE),cpu_pg*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyHostToDevice);
	printf("Copied postfix_guards %lu bytes to %i \n",(int)model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE),cpu_pg);
		if(hipError_t e = hipGetLastError()) {
		  cout << "Error : " << hipGetErrorString(e) << endl;
			assert(0);
		}

	assert(cpu_pg+model->postfix_guards.size() < 63*1024);
	hipMalloc((void **) &pe,model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE));
	hipMemcpy(pe,&model->postfix_effects[0],model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyHostToDevice);
	printf("Copied postfix_effects %lu bytes \n",(int)model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE));

	hipMalloc((void **) &to,model->transition_offsets.size()*sizeof(int));
	hipMemcpy(to,&model->transition_offsets[0],model->transition_offsets.size()*sizeof(int),hipMemcpyHostToDevice);
	printf("Copied transition_offsets %lu bytes \n",(int)model->transition_offsets.size()*sizeof(int));

	if(hipMemcpyToSymbol(HIP_SYMBOL(sizetTransitions_GPU),&model->sizetTransitions,sizeof(unsigned int))) {
		printf("Error copy sizetTransitions  to vram %s\n",hipGetErrorString(hipGetLastError()));
		exit(0);
	}

	unsigned int t = model->initial->size;
	hipMemcpyToSymbol(HIP_SYMBOL("stateSizeGPU"),&t,sizeof(unsigned int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("checkSizeGPU"),&model->checkingSpace,sizeof(unsigned int),0,hipMemcpyHostToDevice);

	initKernel<<<1,1>>>(pg,pe, pr,to);
		if(hipError_t e = hipGetLastError()) {
		  cout << "Error calling kernel: " << hipGetErrorString(e) << endl;
			assert(0);
		}
}

bool checkGPUmemory(myModel * model){
	
	void * tmp = malloc(model->postfix_representation.size()*sizeof(int));
	hipMemcpyFromSymbol(tmp,HIP_SYMBOL("constant"),model->postfix_representation.size()*sizeof(int),cpu_pr*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->postfix_representation[0],model->postfix_representation.size()*sizeof(int)) == 0 );
	free(tmp);

	tmp = malloc(model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE));
	hipMemcpyFromSymbol(tmp,HIP_SYMBOL("constant"),model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE),cpu_pg*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->postfix_guards[0],model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE)) == 0 );
	free(tmp);

	tmp = malloc(model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE));
	hipMemcpy(tmp,pe,model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->postfix_effects[0],model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE)) == 0 );
	free(tmp);

	tmp = malloc(model->transition_offsets.size()*sizeof(int));
	hipMemcpy(tmp,to,model->transition_offsets.size()*sizeof(int),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->transition_offsets[0],model->transition_offsets.size()*sizeof(int)) == 0 );
	free(tmp);

	unsigned int t =0;
	hipMemcpyFromSymbol(&t,HIP_SYMBOL("stateSizeGPU"),sizeof(unsigned int),0,hipMemcpyDeviceToHost);
	assert(t == model->initial->size);

	cout << "GPU Memory Check successfull " << endl;

	return true;
}

void checkGuardsPostfix(unsigned char * stateset,unsigned char * enabledset,unsigned int setSize, myModel * model){
#ifdef _CHECK
	checkGPUmemory(model);
#endif

	if (hipMemcpy(vram,&stateset[0],setSize*model->checkingSpace,hipMemcpyHostToDevice))
		printf("Error copying states to vram %i\n",hipGetLastError());

	unsigned int groups = computeGroups(setSize);
	unsigned int done = 0;
	unsigned int doing = groups*THREADS;
	if (doing > setSize) doing = setSize;

	while (done < setSize) {
		printf("\t done %i computing %i states utilizing %i groups \n",done,doing,groups);
		checkGuardsKernel<<<groups,THREADS>>>(&vram[done*model->checkingSpace], doing);
		if(hipError_t e = hipGetLastError()) {
		  cout << "Error calling kernel: " << hipGetErrorString(e) << endl;
			assert(0);
		}
		doing = setSize-doing-done;
		done += groups*THREADS;
		groups = computeGroups(doing);
	}

	if (hipMemcpy(&enabledset[0],vram,setSize*model->checkingSpace,hipMemcpyDeviceToHost)) {
		assert(0);
	}

}

unsigned int enableEffectsPostfix(unsigned char * childset, unsigned int setSize, myModel * model){
#ifdef _CHECK
	checkGPUmemory(model);
#endif
	if (hipMemcpy(vram,childset,setSize*(model->initial->size+model->sizetTransitions),hipMemcpyHostToDevice)) {
		printf("Error copying %i (%i bytes) children to vram %s\n",setSize,setSize*(model->initial->size+1),hipGetErrorString(hipGetLastError()));
		exit(0);
	}

	unsigned int groups = computeGroups(setSize);

	unsigned int done = 0;
	unsigned int doing = groups*THREADS;
	if (doing > setSize) doing = setSize;

	while (done < setSize) {
		printf("\t done %i computing %i states utilizing %i groups \n",done,doing,groups);
		enableEffectKernel<<<groups,THREADS>>>(&vram[done*(model->initial->size+model->sizetTransitions)], doing );
		if(hipGetLastError()) printf("Error calling kernel %i\n",hipGetLastError());
		done += doing;
		doing = setSize-done;
		cout << " left " << doing << " children" <<endl; 
		groups = computeGroups(doing);
		if (groups == 65000) doing = groups*THREADS;
	}

	if (hipMemcpy(childset,vram,setSize*(model->initial->size+model->sizetTransitions),hipMemcpyDeviceToHost)){
		printf("Error copying %i (%i bytes) children from vram %s\n",setSize,setSize*(model->initial->size+1),hipGetErrorString(hipGetLastError()));
		exit(0);
	}

	return true;
}

unsigned int getHashesGPU(unsigned char * hashes, unsigned int setSize, myModel * model){
#ifdef _CHECK
	checkGPUmemory(model);
#endif
	unsigned int groups = computeGroups(setSize);

	unsigned int done = 0;
	unsigned int doing = groups*THREADS;
	if (doing > setSize) doing = setSize;

	while (done < setSize) {
		printf("\t done %i computing %i states utilizing %i groups \n",done,doing,groups);
		getHashesKernel<<<groups,THREADS>>>(&vram[done*(model->initial->size+model->sizetTransitions)],doing);
		if(hipGetLastError()) printf("Error calling kernel %i\n",hipGetLastError());
		done += doing;
		doing = setSize-done;
		cout << " left " << doing << " children" <<endl; 
		groups = computeGroups(doing);
		if (groups == 65000) doing = groups*THREADS;
	}

	if (hipMemcpy(hashes,vram,setSize*(model->initial->size+model->sizetTransitions),hipMemcpyDeviceToHost)){
		printf("Error copying %i (%i bytes) children from vram %s\n",setSize,setSize*(model->initial->size+1),hipGetErrorString(hipGetLastError()));
		exit(0);
	}

	return setSize;
}

