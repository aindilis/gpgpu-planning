<<<<<<< .mine
/*
 * GPUCheck.cpp
 *
 *  Created on: 05.01.2010
 *      Author: sulewski
 */
#include "GPUCheck.h"

#include "kernels.cu"

__device__ unsigned char * vram; //pointer to GPU memory


__host__ unsigned int computeGroups(unsigned int setSize) {
	unsigned int groups = setSize/THREADS+1;
	if (groups > 65000) {
		printf("\t %i groups splitting \n",groups);
		groups = 65000;
	}

	return groups;
}

void GPUCheck(myModel * model) { //initialize GPU memories

	printf("Initializing GPU \n");

	hipError_t error = hipMalloc((void **) &vram, VRAM);
	if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %s\n","Error allocating vram asfsdf" ,hipGetErrorString(error)); exit(-1); }
	printf("reserved %i MB (%i bytes) on GPU \n",VRAM/1024/1024,VRAM);

	cpu_pr =0;
	hipMemcpyToSymbol(HIP_SYMBOL("pr"),&cpu_pr,sizeof(unsigned int),0,hipMemcpyHostToDevice);
	//hipMalloc((void **) &pr,model->postfix_representation.size()*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL("constant"),&model->postfix_representation[0],model->postfix_representation.size()*sizeof(int),pr,hipMemcpyHostToDevice);
	printf("Copied postfix_representation %lu bytes to %i \n",(int)model->postfix_representation.size()*sizeof(int),pr);

	
	cpu_pg = cpu_pr+model->postfix_representation.size();
	hipMemcpyToSymbol(HIP_SYMBOL("pg"),&cpu_pg,sizeof(unsigned int),0,hipMemcpyHostToDevice);
	//hipMalloc((void **) &pg,model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE)); 
	hipMemcpyToSymbol(HIP_SYMBOL("constant"),&model->postfix_guards[0],model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE),cpu_pg*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyHostToDevice);
	printf("Copied postfix_guards %lu bytes to %i \n",(int)model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE),cpu_pg);
		if(hipError_t e = hipGetLastError()) {
		  cout << "Error : " << hipGetErrorString(e) << endl;
			assert(0);
		}

	assert(cpu_pg+model->postfix_guards.size() < 63*1024);
	hipMalloc((void **) &pe,model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE));
	hipMemcpy(pe,&model->postfix_effects[0],model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyHostToDevice);
	printf("Copied postfix_effects %lu bytes \n",(int)model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE));

	hipMalloc((void **) &to,model->transition_offsets.size()*sizeof(int));
	hipMemcpy(to,&model->transition_offsets[0],model->transition_offsets.size()*sizeof(int),hipMemcpyHostToDevice);
	printf("Copied transition_offsets %lu bytes \n",(int)model->transition_offsets.size()*sizeof(int));

	if(hipMemcpyToSymbol(HIP_SYMBOL(sizetTransitions_GPU),&model->sizetTransitions,sizeof(unsigned int))) {
		printf("Error copy sizetTransitions  to vram %s\n",hipGetErrorString(hipGetLastError()));
		exit(0);
	}

	unsigned int t = model->initial->size;
	hipMemcpyToSymbol(HIP_SYMBOL("stateSizeGPU"),&t,sizeof(unsigned int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("checkSizeGPU"),&model->checkingSpace,sizeof(unsigned int),0,hipMemcpyHostToDevice);

	initKernel<<<1,1>>>(pg,pe, pr,to);
		if(hipError_t e = hipGetLastError()) {
		  cout << "Error calling kernel: " << hipGetErrorString(e) << endl;
			assert(0);
		}
}

bool checkGPUmemory(myModel * model){
	
	void * tmp = malloc(model->postfix_representation.size()*sizeof(int));
	hipMemcpyFromSymbol(tmp,HIP_SYMBOL("constant"),model->postfix_representation.size()*sizeof(int),cpu_pr*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->postfix_representation[0],model->postfix_representation.size()*sizeof(int)) == 0 );
	free(tmp);

	tmp = malloc(model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE));
	hipMemcpyFromSymbol(tmp,HIP_SYMBOL("constant"),model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE),cpu_pg*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->postfix_guards[0],model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE)) == 0 );
	free(tmp);

	tmp = malloc(model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE));
	hipMemcpy(tmp,pe,model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->postfix_effects[0],model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE)) == 0 );
	free(tmp);

	tmp = malloc(model->transition_offsets.size()*sizeof(int));
	hipMemcpy(tmp,to,model->transition_offsets.size()*sizeof(int),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->transition_offsets[0],model->transition_offsets.size()*sizeof(int)) == 0 );
	free(tmp);

	unsigned int t =0;
	hipMemcpyFromSymbol(&t,HIP_SYMBOL("stateSizeGPU"),sizeof(unsigned int),0,hipMemcpyDeviceToHost);
	assert(t == model->initial->size);

	cout << "GPU Memory Check successfull " << endl;

	return true;
}

void checkGuardsPostfix(unsigned char * stateset,unsigned char * enabledset,unsigned int setSize, myModel * model){
#ifdef _CHECK
	checkGPUmemory(model);
#endif

	if (hipMemcpy(vram,&stateset[0],setSize*model->checkingSpace,hipMemcpyHostToDevice))
		printf("Error copying states to vram %i\n",hipGetLastError());

	unsigned int groups = computeGroups(setSize);
	unsigned int done = 0;
	unsigned int doing = groups*THREADS;
	if (doing > setSize) doing = setSize;

	while (done < setSize) {
		printf("\t done %i computing %i states utilizing %i groups \n",done,doing,groups);
		checkGuardsKernel<<<groups,THREADS>>>(&vram[done*model->checkingSpace], doing);
		if(hipError_t e = hipGetLastError()) {
		  cout << "Error calling kernel: " << hipGetErrorString(e) << endl;
			assert(0);
		}
		doing = setSize-doing-done;
		done += groups*THREADS;
		groups = computeGroups(doing);
	}

	if (hipMemcpy(&enabledset[0],vram,setSize*model->checkingSpace,hipMemcpyDeviceToHost)) {
		assert(0);
	}

}

unsigned int enableEffectsPostfix(unsigned char * childset, unsigned int setSize, myModel * model){
#ifdef _CHECK
	checkGPUmemory(model);
#endif
	if (hipMemcpy(vram,childset,setSize*(model->initial->size+model->sizetTransitions),hipMemcpyHostToDevice)) {
		printf("Error copying %i (%i bytes) children to vram %s\n",setSize,setSize*(model->initial->size+1),hipGetErrorString(hipGetLastError()));
		exit(0);
	}

	unsigned int groups = computeGroups(setSize);

	unsigned int done = 0;
	unsigned int doing = groups*THREADS;
	if (doing > setSize) doing = setSize;

	while (done < setSize) {
		printf("\t done %i computing %i states utilizing %i groups \n",done,doing,groups);
		enableEffectKernel<<<groups,THREADS>>>(&vram[done*(model->initial->size+model->sizetTransitions)], doing );
		if(hipGetLastError()) printf("Error calling kernel %i\n",hipGetLastError());
		done += doing;
		doing = setSize-done;
		cout << " left " << doing << " children" <<endl; 
		groups = computeGroups(doing);
		if (groups == 65000) doing = groups*THREADS;
	}

	if (hipMemcpy(childset,vram,setSize*(model->initial->size+model->sizetTransitions),hipMemcpyDeviceToHost)){
		printf("Error copying %i (%i bytes) children from vram %s\n",setSize,setSize*(model->initial->size+1),hipGetErrorString(hipGetLastError()));
		exit(0);
	}

	return true;
}

unsigned int getHashesGPU(unsigned char * hashes, unsigned int setSize, myModel * model){
#ifdef _CHECK
	checkGPUmemory(model);
#endif
	unsigned int groups = computeGroups(setSize);

	unsigned int done = 0;
	unsigned int doing = groups*THREADS;
	if (doing > setSize) doing = setSize;

	while (done < setSize) {
		printf("\t done %i computing %i states utilizing %i groups \n",done,doing,groups);
		getHashesKernel<<<groups,THREADS>>>(&vram[done*(model->initial->size+model->sizetTransitions)],doing);
		if(hipGetLastError()) printf("Error calling kernel %i\n",hipGetLastError());
		done += doing;
		doing = setSize-done;
		cout << " left " << doing << " children" <<endl; 
		groups = computeGroups(doing);
		if (groups == 65000) doing = groups*THREADS;
	}

	if (hipMemcpy(hashes,vram,setSize*(model->initial->size+model->sizetTransitions),hipMemcpyDeviceToHost)){
		printf("Error copying %i (%i bytes) children from vram %s\n",setSize,setSize*(model->initial->size+1),hipGetErrorString(hipGetLastError()));
		exit(0);
	}

	return setSize;
}

=======
/*
 * GPUCheck.cpp
 *
 *  Created on: 05.01.2010
 *      Author: sulewski
 */
#include "GPUCheck.h"

#include "Timer.h"

extern Timer runningTime;


__device__ unsigned char * vram; //pointer to GPU memory


#include "kernels.cu"

#define MAXGROUPS 65000 

void GPUCheck(myModel * model) { //initialize GPU memories

	unsigned int t = model->initial->size;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("stateSizeGPU"),&t,sizeof(unsigned int),0,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("checkSizeGPU"),&model->checkingSpace,sizeof(unsigned int),0,hipMemcpyHostToDevice));

	cpu_pr =0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("pr"),&cpu_pr,sizeof(unsigned int),0,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("constant"),&model->postfix_representation[0],model->postfix_representation.size()*sizeof(int),pr,hipMemcpyHostToDevice));
	cout << runningTime ; printf("Copied postfix_representation %lu bytes to %i \n",(int)model->postfix_representation.size()*sizeof(int),pr);
	
	cpu_pg = cpu_pr+model->postfix_representation.size()*sizeof(int);
	assert(cpu_pg+model->postfix_guards.size() < CONSTANTMEM);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("pg"),&cpu_pg,sizeof(unsigned int),0,hipMemcpyHostToDevice));
	cout << runningTime ; printf("Copying postfix_guards %lu bytes to %i \n",(int)model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE),(unsigned int) cpu_pg*sizeof(POSTFIX_EFFECTS_TYPE));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("constant"),&model->postfix_guards[0],model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE),cpu_pg*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyHostToDevice));
	cout << runningTime ; printf("Copied postfix_guards %lu bytes to %i \n",(int)model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE),(unsigned int) cpu_pg*sizeof(POSTFIX_EFFECTS_TYPE));

	CUDA_SAFE_CALL(hipMalloc((void **) &pe,model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE)));
	CUDA_SAFE_CALL(hipMemcpy(pe,&model->postfix_effects[0],model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyHostToDevice));
	cout << runningTime ; printf("Copied postfix_effects %lu bytes \n",(int)model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE));

	CUDA_SAFE_CALL(hipMalloc((void **) &to,model->transition_offsets.size()*sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(to,&model->transition_offsets[0],model->transition_offsets.size()*sizeof(int),hipMemcpyHostToDevice));
	cout << runningTime ; printf("Copied transition_offsets %lu bytes \n",(int)model->transition_offsets.size()*sizeof(int));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sizetTransitions_GPU),&model->sizetTransitions,sizeof(unsigned int))) ;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(metricTransitionGPU),&model->metric_transition,sizeof(unsigned int))) ;


	//cout<<"copying precondition_predicates"<<endl;
	vector<unsigned int>flat_precond;
	for ( int i = 0 ; i < model->initial->size*16; i++ ) flat_precond.push_back(0);
	map<unsigned int, vector<unsigned int> >::iterator it = model->precondition_predicates.begin();
	while ( it != model->precondition_predicates.end()){
		vector<unsigned int>::iterator it2 = (*it).second.begin();
		if (it2 != (*it).second.end()) flat_precond[(*it).first*2]=flat_precond.size();
		if (it2 != (*it).second.end()) flat_precond[(*it).first*2+1]=(*it).second.size();
		while (it2 != (*it).second.end()){
			flat_precond.push_back(*it2);
			it2++;
		}
		it++;
	}
//	cout << "flat_precon= ";
//	for ( int i = 0 ; i < flat_precond.size() ; i++ ) cout << i << ":" << flat_precond[i] << endl;
//	cout << endl;

	hipMalloc((void **) &preconditions_gpu,flat_precond.size()*sizeof(int));
	hipMemcpy(preconditions_gpu,&flat_precond[0],flat_precond.size()*sizeof(int),hipMemcpyHostToDevice);
	if(hipError_t e = hipGetLastError()) {
		cout << "Error : " << hipGetErrorString(e) << endl;
		assert(0);
	}
	cout << runningTime ;
	printf("Copied precondition_predicates %lu bytes \n",(int)flat_precond.size()*sizeof(int));

	hipError_t error = hipMalloc((void **) &vram, VRAM);
	if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %s\n","Error allocating vram asfsdf" ,hipGetErrorString(error)); exit(-1); }
	cout << runningTime ;
	printf("reserved %i MB (%i bytes) on GPU \n",VRAM/1024/1024,VRAM);


	initKernel<<<1,1>>>(pg,pe, pr,to,preconditions_gpu);
	if(hipError_t e = hipGetLastError()) {
	  cout << "Error calling kernel: " << hipGetErrorString(e) << endl;
		assert(0);
	}
}

bool checkGPUmemory(myModel * model){
	
	void * tmp = malloc(model->postfix_representation.size()*sizeof(int));
	hipMemcpyFromSymbol(tmp,HIP_SYMBOL("constant"),model->postfix_representation.size()*sizeof(int),cpu_pr*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->postfix_representation[0],model->postfix_representation.size()*sizeof(int)) == 0 );
	free(tmp);

	tmp = malloc(model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE));
	hipMemcpyFromSymbol(tmp,HIP_SYMBOL("constant"),model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE),cpu_pg*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->postfix_guards[0],model->postfix_guards.size()*sizeof(POSTFIX_EFFECTS_TYPE)) == 0 );
	free(tmp);

	tmp = malloc(model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE));
	hipMemcpy(tmp,pe,model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->postfix_effects[0],model->postfix_effects.size()*sizeof(POSTFIX_EFFECTS_TYPE)) == 0 );
	free(tmp);

	tmp = malloc(model->transition_offsets.size()*sizeof(int));
	hipMemcpy(tmp,to,model->transition_offsets.size()*sizeof(int),hipMemcpyDeviceToHost);
	assert(memcmp(tmp,&model->transition_offsets[0],model->transition_offsets.size()*sizeof(int)) == 0 );
	free(tmp);

	unsigned int t =0;
	hipMemcpyFromSymbol(&t,HIP_SYMBOL("stateSizeGPU"),sizeof(unsigned int),0,hipMemcpyDeviceToHost);
	assert(t == model->initial->size);
	hipMemcpyFromSymbol(&t,HIP_SYMBOL("pg"),sizeof(unsigned int),0,hipMemcpyDeviceToHost);
	assert(t == cpu_pg);

	cout << "GPU Memory Check successfull " << endl;

	return true;
}

void callKernel(unsigned int setSize,unsigned int elementSize, void(*kernel)(unsigned char * , unsigned int)){
	unsigned int maxElements = MAXGROUPS*THREADS;
	unsigned int toDo = setSize;
	unsigned int done = 0;
	unsigned int groups = 0;

	while (toDo > 0) {
		unsigned int doNow = toDo;
		if (doNow > maxElements) doNow = maxElements;
		groups = doNow / THREADS +1;
		cout << "\t done " << done << " states; computing " << doNow << " states utilizing " << groups << " groups " << endl;
		hipEvent_t evt;
		hipEventCreate(&evt);
		kernel<<<groups,THREADS>>>(&vram[done*elementSize], doNow);
		CUT_CHECK_ERROR("Kernel execution failed");
		hipEventRecord(evt, NULL);
		while (hipEventQuery(evt) == hipErrorNotReady)
		{
			  usleep(500);
		}
		hipEventDestroy(evt);
		done += doNow;
		toDo -= doNow;
	}
	assert (done == setSize);
}

void checkGuardsPostfix(unsigned char * stateset,unsigned char * enabledset,unsigned int setSize, myModel * model){
#ifdef _CHECK
	checkGPUmemory(model);
#endif

	CUDA_SAFE_CALL(hipMemcpy(vram,&stateset[0],setSize*model->checkingSpace,hipMemcpyHostToDevice));

	callKernel(setSize,model->checkingSpace,checkGuardsKernel);

	CUDA_SAFE_CALL(hipMemcpy(&enabledset[0],vram,setSize*model->checkingSpace,hipMemcpyDeviceToHost)); 

}

unsigned int enableEffectsPostfix(unsigned char * childset, unsigned int setSize, myModel * model){
#ifdef _CHECK
	checkGPUmemory(model);
#endif
	CUDA_SAFE_CALL(hipMemcpy(vram,childset,setSize*(model->initial->size+model->sizetTransitions),hipMemcpyHostToDevice)) ;

	callKernel(setSize,model->initial->size+model->sizetTransitions,enableEffectKernel);
if (model->metric_transition != -1)	callKernel(setSize,model->initial->size+model->sizetTransitions,computeMetricKernel);

	CUDA_SAFE_CALL(hipMemcpy(childset,vram,setSize*(model->initial->size+model->sizetTransitions),hipMemcpyDeviceToHost));

	return true;
}

unsigned int getHashesGPU(unsigned char * hashes, unsigned int setSize, myModel * model){
#ifdef _CHECK
	checkGPUmemory(model);
#endif

	callKernel(setSize,model->initial->size+model->sizetTransitions,getHashesKernel);
	CUDA_SAFE_CALL(hipMemcpy(hashes,vram,setSize*(model->initial->size+model->sizetTransitions),hipMemcpyDeviceToHost));
	return setSize;
}

>>>>>>> .r254
